#include <cstdint>
#include <hipcub/hipcub.hpp>

#include "common.cuh"

using namespace hipcub;

namespace common
{

  hipError_t stream_create(hipStream_t &stream, bool blocking_sync)
  {
    hipError_t error = hipStreamCreate(&stream);
    if (error != hipSuccess || !blocking_sync)
    {
      return error;
    }
    hipLaunchAttributeValue policy{};
    policy.syncPolicy = hipSyncPolicyBlockingSync;
    return hipStreamSetAttribute(stream, hipLaunchAttributeSynchronizationPolicy, &policy);
  }

  hipError_t mem_pool_create(hipMemPool_t &mem_pool, int device_id)
  {
    const hipMemPoolProps props = {hipMemAllocationTypePinned, hipMemHandleTypeNone, {hipMemLocationTypeDevice, device_id}};
    HANDLE_RESULT_CUDA(hipMemPoolCreate(&mem_pool, &props));
    uint64_t mem_pool_threshold = UINT64_MAX;
    return hipMemPoolSetAttribute(mem_pool, hipMemPoolAttrReleaseThreshold, &mem_pool_threshold);
  }

} // namespace common

