
#include <hip/hip_runtime_api.h>
#include "common/common.cuh"
#include "panda_interface.cuh"
#include "curve/bn254/config.cuh"
#include "unit/msm/msm_cuda.cuh"
#include "unit/msm/msm_host.cuh"
#include "unit/ntt/fft.cuh"


panda_error panda_get_device_number(int *count)
{
  return static_cast<panda_error>(hipGetDeviceCount(count));
}

panda_error panda_get_device(int *device_id)
{
  return static_cast<panda_error>(hipGetDevice(device_id));
}

panda_error panda_set_device(int device_id)
{
  return static_cast<panda_error>(hipSetDevice(device_id));
}

panda_error panda_stream_create(panda_stream *stream, bool blocking_sync)
{
  return static_cast<panda_error>(common::stream_create(reinterpret_cast<hipStream_t &>(stream->handle), blocking_sync));
}

panda_error panda_stream_wait_event(panda_stream stream, panda_event event)
{
  return static_cast<panda_error>(hipStreamWaitEvent(static_cast<hipStream_t>(stream.handle), static_cast<hipEvent_t>(event.handle)));
}

panda_error panda_stream_sync(panda_stream stream)
{
  return static_cast<panda_error>(hipStreamSynchronize(static_cast<hipStream_t>(stream.handle)));
}

panda_error panda_stream_destroy(panda_stream stream)
{
  return static_cast<panda_error>(hipStreamDestroy(static_cast<hipStream_t>(stream.handle)));
}

panda_error panda_launch_host_fn(panda_stream stream, panda_host_fn fn, void *user_data)
{
  return static_cast<panda_error>(hipLaunchHostFunc(static_cast<hipStream_t>(stream.handle), fn, user_data));
}

panda_error panda_event_create(panda_event *event, bool blocking_sync, bool disable_timing)
{
  int flags = (blocking_sync ? hipEventBlockingSync : hipEventDefault) | (disable_timing ? hipEventDisableTiming : hipEventDefault);
  return static_cast<panda_error>(hipEventCreateWithFlags(reinterpret_cast<hipEvent_t *>(&(event->handle)), flags));
}

panda_error panda_event_record(panda_event event, panda_stream stream)
{
  return static_cast<panda_error>(hipEventRecord(static_cast<hipEvent_t>(event.handle), static_cast<hipStream_t>(stream.handle)));
}

panda_error panda_event_sync(panda_event event)
{
  return static_cast<panda_error>(hipEventSynchronize(static_cast<hipEvent_t>(event.handle)));
}

panda_error panda_event_query(panda_event event)
{
  return static_cast<panda_error>(hipEventQuery(static_cast<hipEvent_t>(event.handle)));
}

panda_error panda_event_destroy(panda_event event)
{
  return static_cast<panda_error>(hipEventDestroy(static_cast<hipEvent_t>(event.handle)));
}

panda_error panda_mem_get_info(size_t *free, size_t *total)
{
  return static_cast<panda_error>(hipMemGetInfo(free, total));
}

panda_error panda_malloc(void **ptr, size_t size)
{
  return static_cast<panda_error>(hipMalloc(ptr, size));
}

panda_error panda_malloc_host(void **ptr, size_t size)
{
  return static_cast<panda_error>(hipHostMalloc(ptr, size));
}

panda_error panda_free(void *ptr)
{
  return static_cast<panda_error>(hipFree(ptr));
}

panda_error panda_free_host(void *ptr)
{
  return static_cast<panda_error>(hipHostFree(ptr));
  }

panda_error panda_host_register(void *ptr, size_t size)
{
  return static_cast<panda_error>(hipHostRegister(ptr, size, hipHostRegisterDefault));
}

panda_error panda_host_unregister(void *ptr)
{
  return static_cast<panda_error>(hipHostUnregister(ptr));
}

panda_error panda_memcpy(void *dst, const void *src, size_t count)
{
  return static_cast<panda_error>(hipMemcpy(dst, src, count, hipMemcpyDefault));
}

panda_error panda_memcpy_async(void *dst, const void *src, size_t count, panda_stream stream)
{
  return static_cast<panda_error>(hipMemcpyAsync(dst, src, count, hipMemcpyDefault, static_cast<hipStream_t>(stream.handle)));
}

panda_error panda_memset(void *ptr, int value, size_t count)
{
  return static_cast<panda_error>(hipMemset(ptr, value, count));
}

panda_error panda_memset_async(void *ptr, int value, size_t count, panda_stream stream)
{
  return static_cast<panda_error>(hipMemsetAsync(ptr, value, count, static_cast<hipStream_t>(stream.handle)));
}

panda_error panda_mem_pool_create(panda_mem_pool *pool, int device_id)
{
  return static_cast<panda_error>(common::mem_pool_create(reinterpret_cast<hipMemPool_t &>(pool->handle), device_id));
}

panda_error panda_mem_pool_destroy(panda_mem_pool pool)
{
  return static_cast<panda_error>(hipMemPoolDestroy(reinterpret_cast<hipMemPool_t>(pool.handle)));
}

panda_error panda_malloc_from_pool_async(void **ptr, size_t size, panda_mem_pool pool, panda_stream stream)
{
  return static_cast<panda_error>(hipMallocFromPoolAsync(ptr, size, static_cast<hipMemPool_t>(pool.handle), static_cast<hipStream_t>(stream.handle)));
}

panda_error panda_free_async(void *ptr, panda_stream stream)
{
  return static_cast<panda_error>(hipFreeAsync(ptr, static_cast<hipStream_t>(stream.handle)));
}

panda_error panda_msm_setup_bn254()
{
  return static_cast<panda_error>(panda_msm_v1::core_msm_setup_bn254());
}

panda_error panda_msm_execute_bn254(const panda_msm_configuration exec_cfg)
{
  return static_cast<panda_error>(panda_msm_v1::core_msm_execute_bn254(exec_cfg));
}

panda_error panda_msm_execute_bn254_host(const panda_msm_configuration exec_cfg)
{
  return static_cast<panda_error>(panda_msm::core_msm_execute_bn254_host(exec_cfg));
}

panda_error panda_msm_tear_down()
{
  return static_cast<panda_error>(panda_msm_v1::core_msm_tear_down());
}

panda_error panda_ntt_setup_bn254(void* input_omega)
{

  return static_cast<panda_error>(panda_ntt::core_ntt_setup_bn254(input_omega));
}

panda_error panda_ntt_execute_bn254(const panda_ntt_configuration exec_cfg)
{
  return static_cast<panda_error>(panda_ntt::core_ntt_execute_bn254(exec_cfg));
}

panda_error panda_ntt_execute_bn254_v1(const panda_ntt_configuration_v1 exec_cfg)
{
  return static_cast<panda_error>(panda_ntt::core_ntt_execute_bn254_v1(exec_cfg));
}

panda_error panda_ntt_tear_down()
{
  return static_cast<panda_error>(panda_ntt::core_ntt_tear_down());
}
